#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <fstream>
#include <vector>
#include <sstream>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//
//                              ALTIPLANO
//
// Prototypes
//__global__ void Kernel(float*,float*,int,int,int,float*,float*,float*,float*,float*);
__global__ void Kernel(float*,float*,int,int,int,float*,float*,float*,float*,float*);
// Host function
using namespace std;
void SaveOutput(float*,int,int,int,string);
int loadData(float*,char*,int);
void LoadNDVI(float**,int,int,int,int,int,int,char*);
int GetDays_h(int,int);
int isLeap_h(int);
int AcumularDias(int,int*,int,int,int);
void AgregarCeros(float*,float*,int*);
int decomposexEst(float*,int,float*,float*,int,int);

__device__ int decompose(float*,int,float*,float*,int);
__device__ int decompose_without_ruido(float*,int,float*,int);
__device__ void applyReplaceZero(float*,int,float*);
__device__ void applyReconstruction(float*,float*,int,int);
__device__ void applyReplaceZeroxEst(float*,int,float*,int);
__device__ void applyReconstructionxEst(float*,float*,int,int,int);
__device__ void eliminateNegative(int,float*);
__device__ int AcumularLluvia(float*,int,float*,int,int,int);
__device__ int GetDays_d(int,int);
__device__ int isLeap_d(int);

__constant__ float lluvia1[1297];  // Arapa
__constant__ float lluvia2[1297];  // Azangaro
__constant__ float lluvia3[1297];  // Capachica
__constant__ float lluvia4[1297];  // Cojata
__constant__ float lluvia5[1297];  // Huancane
__constant__ float lluvia6[1297];  // Lagunillas
__constant__ float lluvia7[1297];  // Lampa
__constant__ float lluvia8[1297];  // Tambopata

__constant__ int diasAcum[128];

int main(int argc, char** argv)
{
printf ("Proceso inicial: Levantando datos de NDVI\n");
//time_t timer1,timer2;
//time(&timer1);
// acumular dias
int day,month,year;
day=1; month=1; year=1999;
int *AcumDias = (int*)malloc(128*sizeof(int));
AcumularDias(1297,AcumDias,day,month,year);
hipMemcpyToSymbol(HIP_SYMBOL(diasAcum),AcumDias,128*sizeof(int));
// cargar datos de lluvia y copio en la memoria constante del device
float* RainDiariaCeros=new float[2048];
float* ruido=new float[1024];
float *lluvia = (float*)malloc(1297*sizeof(float));
char archLluvia[256];
float *ruido1 = (float*)malloc(1024*8*sizeof(float));
float *ruido2 = (float*)malloc(512*8*sizeof(float));
float *ruido3 = (float*)malloc(256*8*sizeof(float));
float *ruido4 = (float*)malloc(128*8*sizeof(float));
// cargo Arapa
strcpy(archLluvia,"D:\\Proyectos\\RecEsp\\RETest\\Arapa.txt");
loadData(lluvia,archLluvia,1297);
hipMemcpyToSymbol(HIP_SYMBOL(lluvia1), lluvia, 1297*sizeof(float));
AgregarCeros(RainDiariaCeros,lluvia,AcumDias);
decomposexEst(RainDiariaCeros,2048,RainDiariaCeros,ruido1,0,0);
decomposexEst(RainDiariaCeros,1024,RainDiariaCeros,ruido2,0,0);
decomposexEst(RainDiariaCeros,512,RainDiariaCeros,ruido3,0,0);
decomposexEst(RainDiariaCeros,256,RainDiariaCeros,ruido4,0,0);
// cargo Azangaro
strcpy(archLluvia,"D:\\Proyectos\\RecEsp\\RETest\\Azangaro.txt");
loadData(lluvia,archLluvia,1297);
hipMemcpyToSymbol(HIP_SYMBOL(lluvia2), lluvia, 1297*sizeof(float));
AgregarCeros(RainDiariaCeros,lluvia,AcumDias);
decomposexEst(RainDiariaCeros,2048,RainDiariaCeros,ruido1,0,1);
decomposexEst(RainDiariaCeros,1024,RainDiariaCeros,ruido2,0,1);
decomposexEst(RainDiariaCeros,512,RainDiariaCeros,ruido3,0,1);
decomposexEst(RainDiariaCeros,256,RainDiariaCeros,ruido4,0,1);
// cargo Capachica
strcpy(archLluvia,"D:\\Proyectos\\RecEsp\\RETest\\Capachica.txt");
loadData(lluvia,archLluvia,1297);
hipMemcpyToSymbol(HIP_SYMBOL(lluvia3), lluvia, 1297*sizeof(float));
AgregarCeros(RainDiariaCeros,lluvia,AcumDias);
decomposexEst(RainDiariaCeros,2048,RainDiariaCeros,ruido1,0,2);
decomposexEst(RainDiariaCeros,1024,RainDiariaCeros,ruido2,0,2);
decomposexEst(RainDiariaCeros,512,RainDiariaCeros,ruido3,0,2);
decomposexEst(RainDiariaCeros,256,RainDiariaCeros,ruido4,0,2);
// cargo Cojata
strcpy(archLluvia,"D:\\Proyectos\\RecEsp\\RETest\\Cojata.txt");
loadData(lluvia,archLluvia,1297);
hipMemcpyToSymbol(HIP_SYMBOL(lluvia4), lluvia, 1297*sizeof(float));
AgregarCeros(RainDiariaCeros,lluvia,AcumDias);
decomposexEst(RainDiariaCeros,2048,RainDiariaCeros,ruido1,0,3);
decomposexEst(RainDiariaCeros,1024,RainDiariaCeros,ruido2,0,3);
decomposexEst(RainDiariaCeros,512,RainDiariaCeros,ruido3,0,3);
decomposexEst(RainDiariaCeros,256,RainDiariaCeros,ruido4,0,3);
// cargo Huancane
strcpy(archLluvia,"D:\\Proyectos\\RecEsp\\RETest\\Huancane.txt");
loadData(lluvia,archLluvia,1297);
hipMemcpyToSymbol(HIP_SYMBOL(lluvia5), lluvia, 1297*sizeof(float));
AgregarCeros(RainDiariaCeros,lluvia,AcumDias);
decomposexEst(RainDiariaCeros,2048,RainDiariaCeros,ruido1,0,4);
decomposexEst(RainDiariaCeros,1024,RainDiariaCeros,ruido2,0,4);
decomposexEst(RainDiariaCeros,512,RainDiariaCeros,ruido3,0,4);
decomposexEst(RainDiariaCeros,256,RainDiariaCeros,ruido4,0,4);
// cargo Lagunillas
strcpy(archLluvia,"D:\\Proyectos\\RecEsp\\RETest\\Lagunillas.txt");
loadData(lluvia,archLluvia,1297);
hipMemcpyToSymbol(HIP_SYMBOL(lluvia6), lluvia, 1297*sizeof(float));
AgregarCeros(RainDiariaCeros,lluvia,AcumDias);
decomposexEst(RainDiariaCeros,2048,RainDiariaCeros,ruido1,0,5);
decomposexEst(RainDiariaCeros,1024,RainDiariaCeros,ruido2,0,5);
decomposexEst(RainDiariaCeros,512,RainDiariaCeros,ruido3,0,5);
decomposexEst(RainDiariaCeros,256,RainDiariaCeros,ruido4,0,5);
// cargo Lampa
strcpy(archLluvia,"D:\\Proyectos\\RecEsp\\RETest\\Lampa.txt");
loadData(lluvia,archLluvia,1297);
hipMemcpyToSymbol(HIP_SYMBOL(lluvia7), lluvia, 1297*sizeof(float));
AgregarCeros(RainDiariaCeros,lluvia,AcumDias);
decomposexEst(RainDiariaCeros,2048,RainDiariaCeros,ruido1,0,6);
decomposexEst(RainDiariaCeros,1024,RainDiariaCeros,ruido2,0,6);
decomposexEst(RainDiariaCeros,512,RainDiariaCeros,ruido3,0,6);
decomposexEst(RainDiariaCeros,256,RainDiariaCeros,ruido4,0,6);
// cargo Tambopata
strcpy(archLluvia,"D:\\Proyectos\\RecEsp\\RETest\\Tambopata.txt");
loadData(lluvia,archLluvia,1297);
hipMemcpyToSymbol(HIP_SYMBOL(lluvia8), lluvia, 1297*sizeof(float));
AgregarCeros(RainDiariaCeros,lluvia,AcumDias);
decomposexEst(RainDiariaCeros,2048,RainDiariaCeros,ruido1,0,7);
decomposexEst(RainDiariaCeros,1024,RainDiariaCeros,ruido2,0,7);
decomposexEst(RainDiariaCeros,512,RainDiariaCeros,ruido3,0,7);
decomposexEst(RainDiariaCeros,256,RainDiariaCeros,ruido4,0,7);
//
float* ruido1_d;
hipMalloc((void**)&ruido1_d, 1024*8*sizeof(float));
hipMemcpy(ruido1_d,ruido1, 1024*8*sizeof(float), hipMemcpyHostToDevice);

float* ruido2_d;
hipMalloc((void**)&ruido2_d, 512*8*sizeof(float));
hipMemcpy(ruido2_d,ruido2, 512*8*sizeof(float), hipMemcpyHostToDevice);

float* ruido3_d;
hipMalloc((void**)&ruido3_d, 256*8*sizeof(float));
hipMemcpy(ruido3_d,ruido3, 256*8*sizeof(float), hipMemcpyHostToDevice);

float* ruido4_d;
hipMalloc((void**)&ruido4_d, 128*8*sizeof(float));
hipMemcpy(ruido4_d,ruido4, 128*8*sizeof(float), hipMemcpyHostToDevice);

delete[] RainDiariaCeros;
delete[] ruido;
free(lluvia);
free(ruido1);
free(ruido2);
free(ruido3);
free(ruido4);
free(AcumDias);
// cargar datos de ndvi y copio en la memoria del device
char archNdvi[256];
strcpy(archNdvi,"D:\\Proyectos\\RecEsp\\RETest\\ndvi99-06.txt");
int X_MAX=1024;
int Y_MAX=1024;
int Z_MAX=128;
int X_MAX2=225;
int Y_MAX2=225;

int agregar=13;  // se agregan 3 bandas ya que el mayor lag de las estaciones es de 3
int size = X_MAX2 * Y_MAX2 * (Z_MAX+agregar) * sizeof(float);
float *NDVI =(float*)malloc(size); // Asigno memoria en el CPU
int lag=0; // indico cero pq cargaremos ndvi desde la primera banda
LoadNDVI(&NDVI,X_MAX,Y_MAX,Z_MAX+agregar,lag,X_MAX2,Y_MAX2,archNdvi);
float* NDVId;
hipMalloc((void**)&NDVId, size);
hipMemcpy(NDVId,NDVI, size, hipMemcpyHostToDevice);
free(NDVI);
// creamos espacio en device para las salidas
int size2 = 225 * 225 * 1297 * sizeof(float);
int sizeSal=1297*sizeof(float);
float* Pd;
float* salNDVI;
hipMalloc((void**)&Pd, size2);
hipMalloc((void**)&salNDVI, sizeSal);
// creamos salidas en el host
float* varios = (float*)malloc(sizeSal);
float* reconst = (float*)malloc(size2);

// ejecuto kernel
dim3 dimBlock(32, 32);
//dim3 dimGrid(X_MAX/32, Y_MAX/32);
dim3 dimGrid(8, 8);
//Kernel<<<dimGrid, dimBlock>>>(NDVId,Pd,X_MAX2,Y_MAX2,Z_MAX,salNDVI,ruido1_d,ruido2_d,ruido3_d,ruido4_d);
Kernel<<<dimGrid, dimBlock>>>(NDVId,Pd,X_MAX2,Y_MAX2,Z_MAX,ruido1_d,ruido2_d,ruido3_d,ruido4_d,salNDVI);
// Transfiero el vector resultante del device al host
hipMemcpy(reconst,Pd,size2,hipMemcpyDeviceToHost);
hipMemcpy(varios,salNDVI,sizeSal,hipMemcpyDeviceToHost);
// Ejecuto proceso para guardar los resultados en un archivo texto
SaveOutput(reconst,225,225,1297,"salida.txt");
//SaveOutput(varios,1,1,1297,"ndvisalida.txt");

char buffer[5];
bool continuar=true;
while (continuar)
{
printf("");
printf("ingrese coordenada x (1-225): ");
fgets( buffer, 5, stdin );
int coordenadaX = atoi(buffer);
coordenadaX=coordenadaX-1;
printf("ingrese coordenada y (1-225): ");
fgets( buffer, 5, stdin );
int coordenadaY = atoi(buffer);
coordenadaY=coordenadaY-1;
	for (int z=0; z<1297; z++) {
	for (int x=0; x<X_MAX2; x++){ // fila
    for (int y=0; y<Y_MAX2; y++) { //columna
	  if(coordenadaX==y && coordenadaY==x)
	  {
	     varios[z]=reconst[z*Y_MAX2*X_MAX2+x*Y_MAX2 + y];
	  }
	}
    }
	}

SaveOutput(varios,1,1,1297,"pixelsolicitado.txt");

printf("");
printf("Se guardo el pixel solicitado !!!");
printf("");
printf("Presionar 9999 para salir:");
fgets( buffer, 5, stdin );
int salir = atoi(buffer);
if(salir==9999) continuar=false;

}

free(reconst);
free(varios);
hipFree(NDVId);
hipFree(ruido1_d);
hipFree(ruido2_d);
hipFree(ruido3_d);
hipFree(ruido4_d);
hipFree(salNDVI);
hipFree(Pd);
printf("Finalizo ...\n");
system("PAUSE");
return 0;
}
//--------------------------------------------------------------------------------------------------------------------------------------
//__global__ void Kernel(float* NDVId, float *Pd,int WidthX,int WidthY,int WidthZ,float* salNDVI,float* RUIDO1,float* RUIDO2,float* RUIDO3,float* RUIDO4)
__global__ void Kernel(float* NDVId, float *Pd,int WidthX,int WidthY,int WidthZ,float* RUIDO1,float* RUIDO2,float* RUIDO3,float* RUIDO4,float* salNDVI)
{
  // Calculate the column index of the Pd element, denote by x
  int x = threadIdx.x + blockIdx.x * blockDim.x;  // columna
  // Calculate the row index of the Pd element, denote by y
  int y = threadIdx.y + blockIdx.y * blockDim.y;  // fila


if(y<225 && x<225)
{
  // encuentro la estaci�n mas cercana
  int cX[8],cY[8]; // coordenadas
  cX[0]=100;cX[1]=92;cX[2]=130;cX[3]=185;cX[4]=140;cX[5]=39;cX[6]=71;cX[7]=208;
  cY[0]=128;cY[1]=103;cY[2]=182;cY[3]=115;cY[4]=136;cY[5]=199;cY[6]=153;cY[7]=26;
  int lag[8];
  lag[0]=5;lag[1]=5;lag[2]=4;lag[3]=5;lag[4]=4;lag[5]=8;lag[6]=6;lag[7]=13;

  float distancia;
  float mindis=0.0;
  int estacion=0;
  for(int i=0;i<8;i++)
  {
	float fact1=cX[i]-x;
	float pot1=pow(fact1,2);
	float fact2=cY[i]-y;
	float pot2=pow(fact2,2);
	distancia=sqrt(pot1+pot2);
	if(i==0)
	{
	  mindis=distancia;
	  estacion=0;
	}
	if(distancia<mindis)
	{
	  mindis=distancia;
	  estacion=i;
	}
  }
  // cargo NDVI considerando el lag de la estacion mas cercana
  float NDVI[2048];  // NDVI deberia ser 128 pero se va a reutilizar mas adelante por eso se dimensiona a 2048
  int ind;
  for (int z=0+lag[estacion]; z<WidthZ+lag[estacion]; z++) {
	 ind=(z*WidthY*WidthX)+(y*WidthY)+(x);
	 NDVI[z-lag[estacion]]=NDVId[ind];
  }

  // acumulo los datos de lluvia de la estacion mas cercana
  int day,month,year;
  day=1;month=1;year=1999;
  float RAIN[2048]; // RAIN deberia ser 128 pero se va a reutilizar mas adelante por eso se dimensiona a 2048
  if(estacion==0) AcumularLluvia(lluvia1,1297,RAIN,day,month,year);
  if(estacion==1) AcumularLluvia(lluvia2,1297,RAIN,day,month,year);
  if(estacion==2) AcumularLluvia(lluvia3,1297,RAIN,day,month,year);
  if(estacion==3) AcumularLluvia(lluvia4,1297,RAIN,day,month,year);
  if(estacion==4) AcumularLluvia(lluvia5,1297,RAIN,day,month,year);
  if(estacion==5) AcumularLluvia(lluvia6,1297,RAIN,day,month,year);
  if(estacion==6) AcumularLluvia(lluvia7,1297,RAIN,day,month,year);
  if(estacion==7) AcumularLluvia(lluvia8,1297,RAIN,day,month,year);

/*
  if(x==100 && y==128)
  {
	for(int z=0;z<50;z++)
	{
	  salNDVI[z]=13.88;
	}
  }
*/


//  Ajuste

   float SUMx = 0;     //sum of x values
   float SUMy = 0;     //sum of y values
   float SUMxy = 0;    //sum of x * y
   float SUMxx = 0;    //sum of x^2
   float slope = 0;    //slope of regression line
   float y_intercept = 0; //y intercept of regression line
   float AVGy = 0;     //mean of y
   float AVGx = 0;     //mean of x

   //calculate various sums
   for (int i = 0; i < 128; i++)
   {
	  //sum of x
	  SUMx = SUMx + NDVI[i];
	  //sum of y
	  SUMy = SUMy + RAIN[i];
	  //sum of squared x*y
	  SUMxy = SUMxy + NDVI[i] * RAIN[i];
	  //sum of squared x
	  SUMxx = SUMxx + NDVI[i] * NDVI[i];
   }

   //calculate the means of x and y
   AVGy = SUMy / 128;
   AVGx = SUMx / 128;

   //slope or a1
   float calc=128 * SUMxx - SUMx*SUMx;
   if(calc<=0.0)
   {
	 slope=0.0;
	 y_intercept=0.0;
   }
   else
   {
	 slope = (128 * SUMxy - SUMx * SUMy) / (128 * SUMxx - SUMx*SUMx);
   //y itercept or a0
   y_intercept = AVGy - slope * AVGx;
   }
   for (int z=0; z<WidthZ; z++) {
	 NDVI[z]=slope*NDVI[z]+y_intercept; // se ajusta los datos de NDVI
   }

	float LD1[64];
	float LD2[32];
	float LD3[16];

	decompose(RAIN,128,RAIN,LD1,0); // descomponemos la lluvia acumulada en tendencia y ruido, la tendencia se guarda nuevamente en el vector RAIN
	decompose(RAIN,64,RAIN,LD2,0);
	decompose(RAIN,32,RAIN,LD3,0);

	decompose_without_ruido(NDVI,128,NDVI,0);
	decompose_without_ruido(NDVI,64,NDVI,0);
	decompose_without_ruido(NDVI,32,NDVI,0);

	for(int i=0;i<16;i++)
	{
	  RAIN[i]=NDVI[i];
	}


	applyReplaceZero(LD3,16,RAIN);
	applyReconstruction(LD3,RAIN,16,0);
	eliminateNegative(32,RAIN);

	applyReplaceZero(LD2,32,RAIN);
	applyReconstruction(LD2,RAIN,32,0);
	eliminateNegative(64,RAIN);

	applyReplaceZero(LD1,64,RAIN);
	applyReconstruction(LD1,RAIN,64,0);
	eliminateNegative(128,RAIN);  // aqui el vector RAIN guarda la lluvia acumulada reconstruida (128 datos)


// hasta aqui tenemos la reconstruccion acumulada de la lluvia almacenada en NAm

// a los datos de lluvia diaria se le agrega ceros
  int cont=0;
  int j=0;
  int contrain=0;
  for(int i=0;i<128;i++)
  {
	cont=0;
	for(int x=0;x<16;x++)
	{
	  cont++;
	  if(cont<=diasAcum[i])
	  { // aqui se agrega ceros a la lluvia diaria y se usa el vector NDVI para guardar la lluvia diaria aumentada
		if(estacion==0) NDVI[j]=lluvia1[contrain];
		if(estacion==1) NDVI[j]=lluvia2[contrain];
		if(estacion==2) NDVI[j]=lluvia3[contrain];
		if(estacion==3) NDVI[j]=lluvia4[contrain];
		if(estacion==4) NDVI[j]=lluvia5[contrain];
		if(estacion==5) NDVI[j]=lluvia6[contrain];
		if(estacion==6) NDVI[j]=lluvia7[contrain];
		if(estacion==7) NDVI[j]=lluvia8[contrain];
		contrain++;
	  }
	  else
	  {
		NDVI[j]=0.0;
	  }
	  j++;
	}
  }
// usaremos el vector NDVI para guardar la tendencia al 4 nivel de descomposicion (128 datos) de la lluvia diaria aumentada con ceros
  decompose_without_ruido(NDVI,2048,NDVI,0);
  decompose_without_ruido(NDVI,1024,NDVI,0);
  decompose_without_ruido(NDVI,512,NDVI,0);
  decompose_without_ruido(NDVI,256,NDVI,0);

  float minN,maxN,minT,maxT;
  minN=1000;
  minT=1000;
  maxN=-1000;
  maxT=-1000;
  for(int i=0;i<128;i++)
  {
	if(RAIN[i]<minN)
	{
	  minN=RAIN[i];
	}
	if(NDVI[i]<minT)
	{
	  minT=NDVI[i];
	}
	if(RAIN[i]>maxN)
	{
	  maxN=RAIN[i];
	}
	if(NDVI[i]>maxT)
	{
	  maxT=NDVI[i];
	}
  }
  float factor,b;

  if(maxN-minN==0.0)
  {
	factor=0.0;
  }
  else
  {
	factor=(maxT-minT)/(maxN-minN);
  }
  b=maxT-(factor*maxN);

  for(int i=0;i<128;i++)
  {
	RAIN[i]=(RAIN[i] * factor)+b;
  }


// reconstruccion

applyReplaceZeroxEst(RUIDO4,128,RAIN,estacion);
applyReconstructionxEst(RUIDO4,RAIN,128,0,estacion);

applyReplaceZeroxEst(RUIDO3,256,RAIN,estacion);
applyReconstructionxEst(RUIDO3,RAIN,256,0,estacion);

applyReplaceZeroxEst(RUIDO2,512,RAIN,estacion);
applyReconstructionxEst(RUIDO2,RAIN,512,0,estacion);

applyReplaceZeroxEst(RUIDO1,1024,RAIN,estacion);
applyReconstructionxEst(RUIDO1,RAIN,1024,0,estacion);

eliminateNegative(2048,RAIN);

// quitar ceros

j=0;
cont=0;
int contar=0;
for(int i=0;i<2048;i++)
{
  contar++;
  if(contar<=diasAcum[j])
  {
	RAIN[cont]=RAIN[i];
	cont++;
  }
  if(contar==16)
  {
	contar=0;
	j++;
  }
}

	for (int z=0; z<1297; z++) {
	  Pd[z*225*225+y*225 + x]= RAIN[z];
	}



}

}
// ------------------------------------------------------------------------------------------------------------------------------------------------
int loadData(float* vector,char* fileName,int num)
{
  int i,count;
  float data;
  FILE *fp;
  fp = fopen( fileName, "r");
  rewind(fp);
  count=0;
  for (i = 0;(fscanf(fp, "%f",&(data)) == 1); i++)
  {
	vector[i]=data;
	count++;
	if(count==num) break;
  }
  fclose(fp);
  return count;
}
//---------------------------------------------------------------------------
__device__ void eliminateNegative(int countNAm,float* NAm)
{
  for(int i=0;i<countNAm;i++)
  {
    if(NAm[i]<0.0)
    {
	  NAm[i]=0.0;
    }
  }
}
//--------------------------------------------------------------------------------------------------------------------------------------
__device__ void applyReplaceZero(float* pvector,int numreg, float* NAm)
{
//  double* vector=NULL;
//  vector=*pvector;
  for(int i=0;i<numreg;i++)
  {
	if(pvector[i]==0.0)
	{
	  NAm[i]=0.0;
	}
  }
//  vector=NULL;
}
//--------------------------------------------------------------------------------------------------------------------------------------
__device__ void applyReplaceZeroxEst(float* pvector,int numreg, float* NAm,int estacion)
{
  int ind=0;
  for(int i=0;i<numreg;i++)
  {
	ind=(estacion*numreg)+i;
	if(pvector[ind]==0.0)
	{
	  NAm[i]=0.0;
	}
  }
}
//--------------------------------------------------------------------------------------------------------------------------------------
void SaveOutput(float *salida,int X_MAX, int Y_MAX, int Z_MAXDiario,string out) {
//  creo matriz de salida
  float*** Matrix3Dfinal = new float**[X_MAX];
  for(int x = 0; x < X_MAX;x++)
  {
    Matrix3Dfinal[x] = new float*[Y_MAX];
    for(int y = 0; y < Y_MAX; y++)
    {
      Matrix3Dfinal[x][y] = new float[Z_MAXDiario];
    }
  }
// paso de vector salida host a la matriz de salida
	for (int z=0; z<Z_MAXDiario; z++) {
	for (int x=0; x<X_MAX; x++){
    for (int y=0; y<Y_MAX; y++) {
	   Matrix3Dfinal[x][y][z]=salida[z*Y_MAX*X_MAX+x*Y_MAX + y];
	}
    }
	}
// guardo matriz salida a un archivo
FILE *stream2=NULL;
string archivo=out;
stream2 = fopen (archivo.c_str(),"w");
rewind(stream2);
string cadena="";
string cadena2;
char* dato;
char* dato2;
for(int z = 0; z < Z_MAXDiario;z++)
{
  for(int x = 0; x < X_MAX;x++)
  {
    cadena="";
    for(int y = 0; y < Y_MAX; y++)
    {
      float valor=Matrix3Dfinal[x][y][z];
      cadena2=std::to_string(long double(valor));
      if(y==Y_MAX-1)
      {
        cadena= cadena + cadena2;
      }
      else
      {
        cadena= cadena + cadena2 + " ";
      }
    }
    dato=new char[cadena.size()+1];
    strcpy(dato,cadena.c_str());
    fprintf(stream2,"%s\n",dato);
	delete(dato);
  }
  cadena="";
  dato2=new char[cadena.size()+1];
  strcpy(dato2,cadena.c_str());
  fprintf(stream2,"%s\n",dato2);
  delete(dato2);
}
fclose(stream2);
// elimino matriz de salida
  for(int x = 0; x < X_MAX; x++)
  {
    for(int y = 0; y < Y_MAX; y++)
    {
      delete[] Matrix3Dfinal[x][y];
    }
    delete[] Matrix3Dfinal[x];
  }
  delete[] Matrix3Dfinal;
  Matrix3Dfinal=0;
}
//---------------------------------------------------------------------------
void LoadNDVI(float** Mp,int X_MAX,int Y_MAX,int Z_MAXDiario,int lag,int X_MAX2,int Y_MAX2,char* arch) {
  float* M=NULL;
  M=*Mp;

  float*** Matrix3Dfinal = new float**[X_MAX2];
  for(int x = 0; x < X_MAX2;x++)
  {
	Matrix3Dfinal[x] = new float*[Y_MAX2];
	for(int y = 0; y < Y_MAX2; y++)
	{
	  Matrix3Dfinal[x][y] = new float[Z_MAXDiario];
	}
  }

  for(int z = 0; z < Z_MAXDiario; z++)
  {
	for(int x = 0; x < X_MAX2;x++)
	{
	  for(int y = 0; y < Y_MAX2; y++)
	  {
		  Matrix3Dfinal[x][y][z]=0.0;
	  }
	}
  }
  FILE* pDato;
  pDato = fopen (arch,"r");
  rewind (pDato);

  int contlag=0;
  float valor=0.0;
  for(int z = 0; z < Z_MAXDiario+lag; z++)
  {
	contlag++;
	for(int x = 0; x < X_MAX2;x++)
	{
      for(int y = 0; y < Y_MAX2; y++)
      {
        fscanf(pDato,"%f", &valor);

	    if(contlag>lag)
		{
		  Matrix3Dfinal[x][y][z-lag]=valor;
		}
      }
    }
  }
  fclose (pDato);


	for (int z=0; z<Z_MAXDiario; z++) {
	for (int x=0; x<X_MAX2; x++){
	for (int y=0; y<Y_MAX2; y++) {
	   M[z*Y_MAX2*X_MAX2+x*Y_MAX2 + y]=Matrix3Dfinal[x][y][z];
	}
	}
	}
 M=0;
}
//---------------------------------------------------------------------------
__device__ int decompose(float* vector,int numreg,float* tendencia,float* ruido,int kindwave)
{
  int i;
  float h0,h1,h2,h3,g0,g1,g2,g3;
  switch (kindwave) {
  case 0: // haar
	h0= 0.7071067814;
	h1= 0.7071067814;
	h2= 0.0;
	h3= 0.0;
	g0= 0.7071067814;
	g1= -0.7071067814;
	g2= 0.0;
	g3= 0.0;
	break;
  case 1: // symmlet2
	h0= 0.482962913;
	h1= 0.836516303;
	h2= 0.224143868;
	h3= -0.129409522;
	g0= -0.129409522;
	g1= -0.224143868;
	g2= 0.836516303;
	g3= -0.482962913;
	break;
  default:
	break;
  }

  int cont=0;
  float aa3T;
  float aa3R;

  aa3T=0.0;
  aa3T+=vector[0]*h2;
  aa3T+=vector[1]*h3;
  aa3T+=vector[numreg-2]*h0;
  aa3T+=vector[numreg-1]*h1;

  aa3R=0.0;
  aa3R+=vector[0]*g2;
  aa3R+=vector[1]*g3;
  aa3R+=vector[numreg-2]*g0;
  aa3R+=vector[numreg-1]*g1;

  float aa3T_back=aa3T;
  float aa3R_back=aa3R;

  for(i=0;i<=numreg-3;i=i+2)
  {
	aa3T=0.0;
	aa3R=0.0;

	aa3T+=vector[i]*h0;
	aa3T+=vector[i+1]*h1;
	aa3T+=vector[i+2]*h2;
	aa3T+=vector[i+3]*h3;

	aa3R+=vector[i]*g0;
	aa3R+=vector[i+1]*g1;
	aa3R+=vector[i+2]*g2;
	aa3R+=vector[i+3]*g3;

	tendencia[cont]=aa3T;
	ruido[cont]=aa3R;
	cont++;
  }

  tendencia[(numreg/2)-1]=aa3T_back;
  ruido[(numreg/2)-1]=aa3R_back;

  return numreg/2;
}
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------
__device__ int decompose_without_ruido(float* vector,int numreg,float* tendencia,int kindwave)
{
  int i;
  float h0,h1,h2,h3;
//  float g0,g1,g2,g3;
  switch (kindwave) {
  case 0: // haar
	h0= 0.7071067814;
	h1= 0.7071067814;
	h2= 0.0;
	h3= 0.0;
//    g0= 0.7071067814;
//    g1= -0.7071067814;
//    g2= 0.0;
//    g3= 0.0;
    break;
  case 1: // symmlet2
    h0= 0.482962913;
    h1= 0.836516303;
    h2= 0.224143868;
    h3= -0.129409522;
//    g0= -0.129409522;
//    g1= -0.224143868;
//    g2= 0.836516303;
//    g3= -0.482962913;
    break;
  default:
    break;
  }

  int cont=0;
  float aa3T;

  aa3T=0.0;
  aa3T+=vector[0]*h2;
  aa3T+=vector[1]*h3;
  aa3T+=vector[numreg-2]*h0;
  aa3T+=vector[numreg-1]*h1;


  float aa3T_back=aa3T;

  for(i=0;i<=numreg-3;i=i+2)
  {
    aa3T=0.0;

	aa3T+=vector[i]*h0;
    aa3T+=vector[i+1]*h1;
    aa3T+=vector[i+2]*h2;
    aa3T+=vector[i+3]*h3;

    tendencia[cont]=aa3T;
    cont++;
  }

  tendencia[(numreg/2)-1]=aa3T_back;

  return numreg/2;
}
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------
__device__  void applyReconstruction(float* pvector,float* NAmm,int numreg,int kindwave)
{
  float h0,h1,h2,h3,g0,g1,g2,g3;
  if(kindwave==0)
  {
    h0= 0.7071067814;
    h1= 0.7071067814;
    h2= 0.0;
    h3= 0.0;
    g0= 0.7071067814;
    g1= -0.7071067814;
    g2= 0.0;
    g3= 0.0;
  }
  else
  {
    h0= 0.482962913;
    h1= 0.836516303;
    h2= 0.224143868;
    h3= -0.129409522;
    g0= -0.129409522;
    g1= -0.224143868;
    g2= 0.836516303;
    g3= -0.482962913;
  }



  int res=0;
  float aa3=0.0;
  int i,j;
  float temp[2048];
  for(i=0;i<numreg;i++)
  {
    temp[i]=NAmm[i];
  }

  aa3=0.0;
  aa3+=temp[0]*h0;
  aa3+=pvector[0]*g0;
  aa3+=temp[numreg-1]*h2;
  aa3+=pvector[numreg-1]*g2;
  NAmm[0]=aa3;

  aa3=0.0;
  aa3+=temp[0]*h1;
  aa3+=pvector[0]*g1;
  aa3+=temp[numreg-1]*h3;
  aa3+=pvector[numreg-1]*g3;
  NAmm[1]=aa3;

  for(i=2;i<2*numreg;i++)
  {
    res=i%2;
	aa3=0.0;
	if(res==0)  // es par
	{
	     j=i-2;
         aa3+=temp[j/2]*h2;
         aa3+=pvector[j/2]*g2;
         aa3+=temp[(j/2)+1]*h0;
         aa3+=pvector[(j/2)+1]*g0;
	}
	else  // es impar
	{
	     j=i-2-1;
         aa3+=temp[j/2]*h3;
         aa3+=pvector[j/2]*g3;
         aa3+=temp[(j/2)+1]*h1;
         aa3+=pvector[(j/2)+1]*g1;
	}
	NAmm[i]=aa3;
  }
}
//--------------------------------------------------------------------------------------------------------------------------------------
__device__  void applyReconstructionxEst(float* pvector,float* NAmm,int numreg,int kindwave,int estacion)
{
  float h0,h1,h2,h3,g0,g1,g2,g3;
  if(kindwave==0)
  {
    h0= 0.7071067814;
    h1= 0.7071067814;
    h2= 0.0;
    h3= 0.0;
    g0= 0.7071067814;
    g1= -0.7071067814;
    g2= 0.0;
    g3= 0.0;
  }
  else
  {
    h0= 0.482962913;
    h1= 0.836516303;
    h2= 0.224143868;
    h3= -0.129409522;
    g0= -0.129409522;
    g1= -0.224143868;
    g2= 0.836516303;
    g3= -0.482962913;
  }

  int res=0;
  float aa3=0.0;
  int i,j;
  float temp[2048];
  for(i=0;i<numreg;i++)
  {
	temp[i]=NAmm[i];
  }

  aa3=0.0;
  aa3+=temp[0]*h0;
  aa3+=pvector[estacion*numreg+0]*g0;
  aa3+=temp[numreg-1]*h2;
  aa3+=pvector[estacion*numreg+(numreg-1)]*g2;
  NAmm[0]=aa3;

  aa3=0.0;
  aa3+=temp[0]*h1;
  aa3+=pvector[estacion*numreg+0]*g1;
  aa3+=temp[numreg-1]*h3;
  aa3+=pvector[estacion*numreg+(numreg-1)]*g3;
  NAmm[1]=aa3;

  for(i=2;i<2*numreg;i++)
  {
	res=i%2;
	aa3=0.0;
	if(res==0)  // es par
	{
		 j=i-2;
		 aa3+=temp[j/2]*h2;
		 aa3+=pvector[estacion*numreg+(j/2)]*g2;
		 aa3+=temp[(j/2)+1]*h0;
		 aa3+=pvector[estacion*numreg+((j/2)+1)]*g0;
	}
	else  // es impar
	{
		 j=i-2-1;
		 aa3+=temp[j/2]*h3;
		 aa3+=pvector[estacion*numreg+(j/2)]*g3;
		 aa3+=temp[(j/2)+1]*h1;
		 aa3+=pvector[estacion*numreg+((j/2)+1)]*g1;
	}
	NAmm[i]=aa3;
  }
}
//--------------------------------------------------------------------------------------------------------------------------------------
__device__ int AcumularLluvia(float* lluvia,int numdatos,float* rainAcum,int day,int month,int year)
{
  int decadal;
  if(day<=10) decadal=1;
  if(day>10 && day<=20) decadal=2;
  if(day>20) decadal=3;
  // acumulo la lluvia en decadales
  float acum=0.0;
  int contDias=0;
  int dia=day;
  int contAcum=0;
  int TotalDaysThisDecadal;
  for(int i=0;i<numdatos;i++)
  {
	acum=acum+lluvia[i];
	contDias++;
	if(decadal==1 && dia==10){rainAcum[contAcum]=acum;decadal++;dia=0;acum=0;contAcum++;contDias=0;}
	if(decadal==2 && dia==10){rainAcum[contAcum]=acum;decadal++;dia=0;acum=0;contAcum++;contDias=0;}
	if(decadal==3){TotalDaysThisDecadal=GetDays_d(month,year);}
	if(decadal==3 && dia==TotalDaysThisDecadal)
	{
	  rainAcum[contAcum]=acum;
	  decadal=1;
	  dia=0;
	  acum=0.0;
	  contAcum++;
	  if(month==12)
	  {month=1;year++;}
	  else
	  {month++;}
	  contDias=0;
	}
	dia++;
  }
  return contAcum;
}
//---------------------------------------------------------------------------
int AcumularDias(int numdatos,int* diasAcum,int day,int month,int year)
{
  int decadal;
  if(day<=10) decadal=1;
  if(day>10 && day<=20) decadal=2;
  if(day>20) decadal=3;
  // acumulo la lluvia en decadales
  int contDias=0;
  int dia=day;
  int contAcum=0;
  int TotalDaysThisDecadal;
  for(int i=0;i<numdatos;i++)
  {
	contDias++;
	if(decadal==1 && dia==10){diasAcum[contAcum]=contDias;decadal++;dia=0;contAcum++;contDias=0;}
	if(decadal==2 && dia==10){diasAcum[contAcum]=contDias;decadal++;dia=0;contAcum++;contDias=0;}
	if(decadal==3){TotalDaysThisDecadal=GetDays_h(month,year);}
	if(decadal==3 && dia==TotalDaysThisDecadal)
	{
	  diasAcum[contAcum]=contDias;
	  decadal=1;
	  dia=0;
	  contAcum++;
	  if(month==12)
	  {month=1;year++;}
	  else
	  {month++;}
	  contDias=0;
	}
	dia++;
  }
  return contAcum;
}
//---------------------------------------------------------------------------
int GetDays_h(int month,int year)
{
int dato;
switch (month) {
  case 1 :
	return 11;
  case 2 :
	if(isLeap_h(year)){dato=9;}
	else{dato=8;}
	return dato;
  case 3 :
	return 11;
  case 4 :
	return 10;
  case 5 :
	return 11;
  case 6 :
	return 10;
  case 7 :
	return 11;
  case 8 :
	return 11;
  case 9 :
	return 10;
  case 10 :
	return 11;
  case 11 :
	return 10;
  case 12 :
	return 11;
}
return 0;
}
//---------------------------------------------------------------------------
__device__ int GetDays_d(int month,int year)
{
int dato;
switch (month) {
  case 1 :
	return 11;
  case 2 :
	if(isLeap_d(year)){dato=9;}
	else{dato=8;}
	return dato;
  case 3 :
	return 11;
  case 4 :
	return 10;
  case 5 :
	return 11;
  case 6 :
	return 10;
  case 7 :
	return 11;
  case 8 :
	return 11;
  case 9 :
	return 10;
  case 10 :
	return 11;
  case 11 :
	return 10;
  case 12 :
	return 11;
}
return 0;
}
//---------------------------------------------------------------------------
int isLeap_h(int year)
{
	return ((year % 4 == 0 && year % 100 != 0) || year % 400 == 0);
}
//---------------------------------------------------------------------------
__device__ int isLeap_d(int year)
{
	return ((year % 4 == 0 && year % 100 != 0) || year % 400 == 0);
}
//---------------------------------------------------------------------------
void AgregarCeros(float* RainDiariaCeros,float* lluvia,int* diasAcum)
{
  int cont=0;
  int j=0;
  int contrain=0;
  for(int i=0;i<128;i++)
  {
	cont=0;
	for(int x=0;x<16;x++)
	{
	  cont++;
	  if(cont<=diasAcum[i])
	  { // aqui se agrega ceros a la lluvia diaria y se usa el vector NDVI para guardar la lluvia diaria aumentada
		RainDiariaCeros[j]=lluvia[contrain];
		contrain++;
	  }
	  else
	  {
		RainDiariaCeros[j]=0.0;
	  }
	  j++;
	}
  }
}
//---------------------------------------------------------------------------
int decomposexEst(float* vector,int numreg,float* tendencia,float* ruido,int kindwave,int est)
{
  int i;
  float h0,h1,h2,h3,g0,g1,g2,g3;
  switch (kindwave) {
  case 0: // haar
	h0= 0.7071067814;
	h1= 0.7071067814;
	h2= 0.0;
	h3= 0.0;
	g0= 0.7071067814;
	g1= -0.7071067814;
	g2= 0.0;
	g3= 0.0;
	break;
  case 1: // symmlet2
	h0= 0.482962913;
	h1= 0.836516303;
	h2= 0.224143868;
	h3= -0.129409522;
	g0= -0.129409522;
	g1= -0.224143868;
	g2= 0.836516303;
	g3= -0.482962913;
	break;
  default:
	break;
  }

  int cont=0;
  float aa3T;
  float aa3R;

  aa3T=0.0;
  aa3T+=vector[0]*h2;
  aa3T+=vector[1]*h3;
  aa3T+=vector[numreg-2]*h0;
  aa3T+=vector[numreg-1]*h1;

  aa3R=0.0;
  aa3R+=vector[0]*g2;
  aa3R+=vector[1]*g3;
  aa3R+=vector[numreg-2]*g0;
  aa3R+=vector[numreg-1]*g1;

  float aa3T_back=aa3T;
  float aa3R_back=aa3R;

  for(i=0;i<=numreg-3;i=i+2)
  {
	aa3T=0.0;
	aa3R=0.0;

	aa3T+=vector[i]*h0;
	aa3T+=vector[i+1]*h1;
	aa3T+=vector[i+2]*h2;
	aa3T+=vector[i+3]*h3;

	aa3R+=vector[i]*g0;
	aa3R+=vector[i+1]*g1;
	aa3R+=vector[i+2]*g2;
	aa3R+=vector[i+3]*g3;

	tendencia[cont]=aa3T;
	ruido[(est*(numreg/2))+cont]=aa3R;
	cont++;
  }

  tendencia[(numreg/2)-1]=aa3T_back;
  ruido[(est*(numreg/2))+((numreg/2)-1)]=aa3R_back;

  return numreg/2;
}
//---------------------------------------------------------------------------